/**
 * cuda_delta_knn.cu
 * block loading delta calculation.
 * system('nvcc -ptx -m 64 -arch sm_35 jrc3_cuda_rho.cu')
 * J. James Jun, Flatiron Institute, 2018 July 5
*/

#include <hip/hip_runtime.h>
// #include "cublas_v2.h"
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define NC (3*20) // number of Channels
#define CHUNK 16 //previously defined as CHUNK
#define SINGLE_INF (3.402E+38)

__global__ void cuda_delta_knn(float *D, unsigned int *N, const float *B, const float *A, const float *R_B, const float *R_A, const int *vnConst){
    
    int nB = vnConst[0];
    int nA = vnConst[1];
    int nC = vnConst[2];

    int tx = threadIdx.x;
    int iA = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNK + tx;
    int nThreads = blockDim.x; // must be less than NTHREADS

    __shared__ float sA[NC][CHUNK], sR_A[CHUNK]; // march through A
    __shared__ float sD[NTHREADS][CHUNK]; // count then divide later    
    __shared__ unsigned int sN[NTHREADS][CHUNK]; 
    
    
    // initialize
    if (tx < CHUNK){         
        if (iA < nA){
            int iA_ = tx;
            for (int iC=0; iC<nC; ++iC) sA[iC][tx] = A[iC + iA*nC]; // copy A->sA
            sR_A[iA_] = R_A[iA]; // copy R_A->sR_A
            for (int iB_=0; iB_<nThreads; ++iB_) sD[iB_][iA_] = SINGLE_INF; // sD = inf
        }
    }
    __syncthreads();        

    
    // Search min distance having a greater rho
    for (int iB=tx; iB<nB; iB+=nThreads){                
        // compute distance
        float dist_[CHUNK];
        for (int iA_=0; iA_<CHUNK; ++iA_) dist_[iA_] = 0.0f;        
        for (int iC=0; iC<nC; ++iC){
            float b_ = B[iC + iB * nC];
            for (int iA_=0; iA_<CHUNK; ++iA_){
                float d_ = b_ - sA[iC][iA_];
                dist_[iA_] += (d_ * d_);
            }            
        }
        
        // Compare the index and distance
        float rb_ = R_B[iB];
        int iB_ = tx;
        for (int iA_=0; iA_<CHUNK; ++iA_){   
            if (rb_ > sR_A[iA_] && dist_[iA_] < sD[iB_][iA_]){
                sD[iB_][iA_] = dist_[iA_];
                sN[iB_][iA_] = iB;
            }
        }
    } // while    
    __syncthreads();    
    
    
    // final count
    if (tx < CHUNK){        
        if (iA < nA){
            int iA_ = tx;
            float dmin_ = SINGLE_INF;
            unsigned int imin_ = iA; // point to self initially
            for (int iB_=0; iB_<nThreads; ++iB_){                
                if (sD[iB_][iA_] < dmin_){
                    dmin_ = sD[iB_][iA_];
                    imin_ = sN[iB_][iA_];
                }
            }
            D[iA] = sqrtf(ABS(dmin_));
            N[iA] = imin_ + 1; //Matlab index output
        }
    }
} // func